#include "hip/hip_runtime.h"
/*
Created by Zebulun Arendsee.
March 26, 2013

Modified by Will Landau.
June 30, 2013
will-landau.com
landau@iastate.edu

This program implements a MCMC algorithm for the following hierarchical
model:

y_k     ~ Poisson(n_k * theta_k)     k = 1, ..., K
theta_k ~ Gamma(a, b)
a       ~ Unif(0, a0)
b       ~ Unif(0, b0) 

We let a0 and b0 be arbitrarily large.

Arguments:
    1) input filename
        With two space delimited columns holding integer values for
        y and float values for n.
    2) number of trials (1000 by default)

Output: A comma delimited file containing a column for a, b, and each
theta. All output is written to stdout.

Example dataset:

$ head -3 data.txt
4 0.91643
23 3.23709
7 0.40103

Example of compilation and execution:

$ nvcc gibbs_metropolis.cu -o gibbs
$ ./gibbs mydata.txt 2500 > output.csv
$

This code borrows from the nVidia developer zone documentation, 
specifically http://docs.nvidia.com/cuda/hiprand/index.html#topic_1_2_1
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <thrust/reduce.h>
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>

#define PI 3.14159265359f
#define THREADS_PER_BLOCK 64

#define CUDA_CALL(x) {if((x) != hipSuccess){ \
  printf("CUDA error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}} 

#define CURAND_CALL(x) {if((x) != HIPRAND_STATUS_SUCCESS) { \
  printf("Error at %s:%d\n",__FILE__,__LINE__); \
  printf("  %s\n", hipGetErrorString(hipGetLastError())); \
  exit(EXIT_FAILURE);}}

__host__ void load_data(int argc, char **argv, int *K, int **y, float **n);

__host__ float sample_a(float a, float b, int K, float sum_logs);
__host__ float sample_b(float a, int K, float flat_sum);

__host__ float rnorm();
__host__ float rgamma(float a, float b);

__device__ float rgamma(hiprandState *state, int id, float a, float b);

__global__ void sample_theta(hiprandState *state, float *theta, float *log_theta, 
                             int *y, float *n, float a, float b, int K);
__global__ void setup_kernel(hiprandState *state, unsigned int seed, int);


int main(int argc, char **argv){

  hiprandState *devStates;
  float a, b, flat_sum, sum_logs, *n, *dev_n, *dev_theta, *dev_log_theta;
  int i, K, *y, *dev_y, nBlocks, trials = 1000;

  if(argc > 2)
    trials = atoi(argv[2]);

  load_data(argc, argv, &K, &y, &n);


  /*------ Allocate memory -----------------------------------------*/

  CUDA_CALL(hipMalloc((void **)&dev_y, K * sizeof(int)));
  CUDA_CALL(hipMemcpy(dev_y, y, K * sizeof(int), 
            hipMemcpyHostToDevice));

  CUDA_CALL(hipMalloc((void **)&dev_n, K * sizeof(float)));
  CUDA_CALL(hipMemcpy(dev_n, n, K * sizeof(float), 
            hipMemcpyHostToDevice));

  /* Allocate space for theta and log_theta on device and host */
  CUDA_CALL(hipMalloc((void **)&dev_theta, K * sizeof(float)));
  CUDA_CALL(hipMalloc((void **)&dev_log_theta, K * sizeof(float)));

  /* Allocate space for random states on device */
  CUDA_CALL(hipMalloc((void **)&devStates, K * sizeof(hiprandState)));


  /*------ Setup random number generators (one per thread) ---------*/

  nBlocks = (K + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
  setup_kernel<<<nBlocks, THREADS_PER_BLOCK>>>(devStates, 0, K);


  /*------ MCMC ----------------------------------------------------*/
    
  printf("alpha, beta\n");

  /* starting values of hyperparameters */
  a = 20; 
  b = 1; 
    
  /* Steps of MCMC */  
  for(i = 0; i < trials; i++){    
    sample_theta<<<nBlocks, THREADS_PER_BLOCK>>>(devStates, dev_theta, dev_log_theta,
                                                 dev_y, dev_n, a, b, K);

    /* Make iterators for thetas and log thetas. */
    thrust::device_ptr<float> theta(dev_theta);
    thrust::device_ptr<float> log_theta(dev_log_theta);
    
    /* Compute pairwise sums of thetas and log_thetas. */
    flat_sum = thrust::reduce(theta, theta + K);
    sum_logs = thrust::reduce(log_theta, log_theta + K);
  
    /* Sample hyperparameters. */
    a = sample_a(a, b, K, sum_logs);
    b = sample_b(a, K, flat_sum);
    
    /* print hyperparameters. */
    printf("%f, %f\n", a, b); 
  }

  /*------ Free Memory -------------------------------------------*/

  free(y);
  free(n);

  CUDA_CALL(hipFree(devStates));
  CUDA_CALL(hipFree(dev_theta));
  CUDA_CALL(hipFree(dev_log_theta));
  CUDA_CALL(hipFree(dev_y));
  CUDA_CALL(hipFree(dev_n));

  return EXIT_SUCCESS;
}


/*
 *  Read in data.
 */

__host__ void load_data(int argc, char **argv, int *K, int **y, float **n){
  int k;
  char line[128];
  FILE *fp;    
    
  if(argc > 1){
    fp = fopen(argv[1], "r");
  } else {
    printf("Please provide input filename\n");
    exit(EXIT_FAILURE);
  }

  if(fp == NULL){
    printf("Cannot read file \n");
    exit(EXIT_FAILURE);
  }

  *K = 0;
  while( fgets (line, sizeof line, fp) != NULL )
    (*K)++; 

  rewind(fp);

  *y = (int*) malloc((*K) * sizeof(int));
  *n = (float*) malloc((*K) * sizeof(float)); 
  
  for(k = 0; k < *K; k++)
    fscanf(fp, "%d %f", *y + k, *n + k);    
 
  fclose(fp);
}


/*
 *  Metropolis algorithm for producing random a values. 
 *  The proposal distribution in normal with a variance that
 *  is adjusted at each step.
 */
 
__host__ float sample_a(float a, float b, int K, float sum_logs){

  static float sigma = 2;
  float U, log_acceptance_ratio, proposal = rnorm() * sigma + a;

  if(proposal <= 0) 
    return a;

  log_acceptance_ratio = (proposal - a) * sum_logs +
                         K * (proposal - a) * log(b) -
                         K * (lgamma(proposal) - lgamma(a));

  U = rand() / float(RAND_MAX);

  if(log(U) < log_acceptance_ratio){
    sigma *= 1.1;
    return proposal;
  } else {
    sigma /= 1.1;
    return a;
  }
}


/*
 *  Sample b from a gamma distribution.
 */

__host__ float sample_b(float a, int K, float flat_sum){

  float hyperA = K * a + 1;
  float hyperB = flat_sum;
  return rgamma(hyperA, hyperB);
}


/* 
 *  Box-Muller Transformation: Generate one standard normal variable.
 *
 *  This algorithm can be more efficiently used by producing two
 *  random normal variables. However, for the CPU, much faster
 *  algorithms are possible (e.g. the Ziggurat Algorithm);
 *
 *  This is actually the algorithm chosen by NVIDIA to calculate
 *  normal random variables on the GPU.
 */
 
__host__ float rnorm(){

  float U1 = rand() / float(RAND_MAX);
  float U2 = rand() / float(RAND_MAX);
  float V1 = sqrt(-2 * log(U1)) * cos(2 * PI * U2);
  /* float V2 = sqrt(-2 * log(U2)) * cos(2 * PI * U1); */
  return V1;
}


/*
 *  See device rgamma function. This is probably not the
 *   fastest way to generate random gamma variables on a CPU.
 */
 
__host__ float rgamma(float a, float b){

  float d = a - 1.0 / 3;
  float Y, U, v;

  while(1){
    Y = rnorm();
    v = pow((1 + Y / sqrt(9 * d)), 3);

    /* Necessary to avoid taking the log of a negative number later. */
    if(v <= 0) 
      continue;
        
    U = rand() / float(RAND_MAX);

    /* Accept the sample under the following condition. 
       Otherwise repeat loop. */
    if(log(U) < 0.5 * pow(Y,2) + d * (1 - v + log(v)))
            return d * v / b;
  }
}


/* 
 *  Generate a single Gamma distributed random variable by the Marsoglia 
 *  algorithm (George Marsaglia, Wai Wan Tsang; 2001).
 *
 *  Zeb chose this algorithm because it has a very high acceptance rate (>96%),
 *  so this while loop will usually only need to run a few times. Many other 
 *  algorithms, while perhaps faster on a CPU, have acceptance rates on the 
 *  order of 50% (very bad in a massively parallel context).
 */

__device__ float rgamma(hiprandState *state, int id, float a, float b){

  float d = a - 1.0 / 3;
  float Y, U, v;

  while(1){   
    Y = hiprand_normal(&state[id]);
    v = pow((1 + Y / sqrt(9 * d)), 3);

    /* Necessary to avoid taking the log of a negative number later. */
    if(v <= 0) 
      continue;
        
    U = hiprand_uniform(&state[id]);

    /* Accept the sample under the following condition. 
       Otherwise repeat loop. */
    if(log(U) < 0.5 * pow(Y,2) + d * (1 - v + log(v)))
      return d * v / b;
  }
}


/*
 *  Sample each theta from the appropriate gamma distribution
 */
 
__global__ void sample_theta(hiprandState *state, 
                             float *theta, float *log_theta, int *y, float *n, 
                             float a, float b, int K){
                             
  int id = threadIdx.x + blockIdx.x * blockDim.x;
  float hyperA, hyperB;
    
  if(id < K){
    hyperA = a + y[id];
    hyperB = b + n[id];
    theta[id] = rgamma(state, id, hyperA, hyperB);
    log_theta[id] = log(theta[id]);
  }
}


/* 
 *  Initialize GPU random number generators 
 */
 
__global__ void setup_kernel(hiprandState *state, unsigned int seed, int K){

  int id = threadIdx.x + blockIdx.x * blockDim.x;
  
  if(id < K)
    hiprand_init(seed, id, 0, &state[id]);
}